// compiler flags:
// nvcc -std=c++11 -arch=sm_61 -shared ani_op.cc.cu kernel.cu -o ani.so ${TF_CFLAGS[@]} ${TF_LFLAGS[@]} -I ~/Code/cub-1.8.0/ -Xcompiler -fPIC -O3 -D GOOGLE_CUDA=1 -I /usr/local --expt-relaxed-constexp

#define EIGEN_USE_GPU // do *not* remove, this is used by the tf/eigen headers to define GpuDevice types
// #define GOOGLE_CUDA // define using -D GOOGLE_CUDA 1 on the c++ level instead

// #include "ani_op.h"

#include "tensorflow/core/framework/op_kernel.h"
#include "tensorflow/core/framework/op.h"
#include "tensorflow/core/framework/shape_inference.h"
#include "tensorflow/core/framework/types.h"
#include "tensorflow/core/util/cuda_kernel_helper.h"

#include <chrono>

#include "kernel.cuh"

using namespace tensorflow;

using CPUDevice = Eigen::ThreadPoolDevice;
using GPUDevice = Eigen::GpuDevice;

REGISTER_OP("Ani")
  .Input("xs: float32")
  .Input("ys: float32")
  .Input("zs: float32")
  .Input("as: int32")
  .Input("mos: int32") // mol offsets
  .Input("macs: int32") // mol atom counts
  .Input("sis: int32") // scatter_idxs
  .Input("acs: int32") // atom counts of size 4 (HOST MEMORY)
  .Output("h_feat: float32")
  .Output("c_feat: float32")
  .Output("n_feat: float32")
  .Output("o_feat: float32")
  .SetShapeFn([](::tensorflow::shape_inference::InferenceContext* c) {
    // the output shapes are determined by the number of elements in acs
    // c->set_output(0, c->input(0));
    // c->set_output(0, c->input(0));
    // c->set_output(0, c->input(0));
    // c->set_output(0, c->input(0));
    return Status::OK();
  });


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}   

class AniOp : public OpKernel {

 public:
  explicit AniOp(OpKernelConstruction* context) : OpKernel(context) {
    // empty constructor
  }

  void Compute(OpKernelContext* context) override {


    // hipDeviceSynchronize();
    auto start = std::chrono::high_resolution_clock::now();

    // Grab the input tensors
    const Tensor& input_Xs   = context->input(0);
    const Tensor& input_Ys   = context->input(1);
    const Tensor& input_Zs   = context->input(2);
    const Tensor& input_As   = context->input(3);
    const Tensor& input_MOs  = context->input(4);
    const Tensor& input_MACs = context->input(5);
    const Tensor& input_SIs  = context->input(6);
    const Tensor& input_ACs  = context->input(7); // HOST

    long long total_num_atoms = input_Xs.shape().num_elements();
    long long n_mols = input_MOs.shape().num_elements();
   
    const GPUDevice &d = context->eigen_device<GPUDevice>();

    Tensor* X_feat_H = nullptr;
    Tensor* X_feat_C = nullptr;
    Tensor* X_feat_N = nullptr;
    Tensor* X_feat_O = nullptr;
 
    const int *acs = input_ACs.flat<int>().data(); // safe since we declare this to be on the host.

    OP_REQUIRES_OK(context, context->allocate_output(
      "h_feat",
      TensorShape({acs[0]*384}),
      &X_feat_H)
    );
    OP_REQUIRES_OK(context, context->allocate_output(
      "c_feat",
      TensorShape({acs[1]*384}),
      &X_feat_C)
    );
    OP_REQUIRES_OK(context, context->allocate_output(
      "n_feat",
      TensorShape({acs[2]*384}),
      &X_feat_N)
    );
    OP_REQUIRES_OK(context, context->allocate_output(
      "o_feat",
      TensorShape({acs[3]*384}),
      &X_feat_O)
    );

    gpuErrchk(hipMemsetAsync(X_feat_H->flat<float>().data(), 0, acs[0]*384*sizeof(int), d.stream()));
    gpuErrchk(hipMemsetAsync(X_feat_C->flat<float>().data(), 0, acs[1]*384*sizeof(int), d.stream()));
    gpuErrchk(hipMemsetAsync(X_feat_N->flat<float>().data(), 0, acs[2]*384*sizeof(int), d.stream()));
    gpuErrchk(hipMemsetAsync(X_feat_O->flat<float>().data(), 0, acs[3]*384*sizeof(int), d.stream()));

    if(n_mols > 0) {
      featurize<<<n_mols, 32, 0, d.stream()>>>(
        input_Xs.flat<float>().data(),
        input_Ys.flat<float>().data(),
        input_Zs.flat<float>().data(),
        input_As.flat<int>().data(),
        input_MOs.flat<int>().data(),
        input_MACs.flat<int>().data(),
        n_mols,
        input_SIs.flat<int>().data(),
        X_feat_H->flat<float>().data(),
        X_feat_C->flat<float>().data(),
        X_feat_N->flat<float>().data(),
        X_feat_O->flat<float>().data()
      );
      gpuErrchk(hipPeekAtLastError());
    } else {
      std::cout << "Empty mol" << std::endl;
    }
    



  }
};

REGISTER_KERNEL_BUILDER(Name("Ani").Device(DEVICE_GPU).HostMemory("acs"), AniOp);